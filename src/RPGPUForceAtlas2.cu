#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <chrono>
#include <algorithm>
#include "time.h"
#include "DataStore.hpp"
#include <vector>
#include <cmath>
#include "RPGPUForceAtlas2.hpp"
#include "RPBHFA2LaunchParameters.cuh"
#include "RPBHKernels.cuh"
#include "RPFA2Kernels.cuh"
#include <iostream>

namespace RPGraph
{
    CUDAForceAtlas2::CUDAForceAtlas2(GraphLayout &layout, bool use_barneshut,
                                     bool strong_gravity, float gravity,
                                     float scale, int max_iterations,std::vector<std::vector<int>> &externa_polygonX,float maximum_area)
    : ForceAtlas2(layout, use_barneshut, strong_gravity, gravity, scale,  max_iterations)
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            fprintf(stderr, "error: No CUDA devices found.\n");
            exit(EXIT_FAILURE);
        }

        // Host initialization and setup //
        nbodies = layout.graph.num_nodes();
        nedges  = layout.graph.num_edges();
        h_points = layout.getPolygonPoints();
        body_pos = (float2 *)malloc(sizeof(float2) * layout.graph.num_nodes());
        body_mass = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        sources  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        targets  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        fx       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fx_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        Cy=layout.getCy();
        Cx=layout.getCx();
        max_iterations2= max_iterations;
        DataStore& dataStore = DataStore::getInstance();
        num_points=dataStore.polygons[layout.polygonId].size();
        num_polygons=dataStore.number_of_polygons;
        std::cout<<"number of the points"<<num_points<<std::endl;
        std::cout<<"number of the polygons"<<num_polygons<<std::endl;
        std::cout<<"number of the nodes"<<nbodies<<std::endl;
         std::cout<<"polygon ID"<<layout.polygonId<<std::endl;
    // Calculate effective distances for each corner
    float effective_d[num_points];
   
    for (int k = 0; k < num_points; k++) {
        float corner_x = h_points[ 2 * k];
        float corner_y = h_points[1 + 2 * k];
        float corner_x_next = h_points[1+( 1+2 * k)];
        float corner_y_next = h_points[1+1+(1 + 2 * k)];
        float dx_next = corner_x_next - corner_x;
        float dy_next = corner_y_next- corner_y;
        float d_next=sqrtf(dx_next * dx_next + dy_next * dy_next) ;
        float corner_x_rev = 0;
        float corner_y_rev = 0;
         if(k==0)
        {
             corner_x_rev = h_points[ num_points-2];
             corner_y_rev = h_points[num_points-1];
        }
        else
        {
             corner_x_rev = h_points[ 2-2 * k];
             corner_y_rev = h_points[2-1 + 2 * k];
        }
        float dx_rev = corner_x_rev - corner_x;
        float dy_rev = corner_y_rev - corner_y;
        float d_rev=sqrtf(dx_rev * dx_rev + dy_rev * dy_rev) ;
        float dxC=corner_x-Cx ;
        float dyC=corner_y-Cy ;
        float dC=sqrtf(dxC * dxC + dyC * dyC) ;
        if(dC>max_distance_corner)
        max_distance_corner=dC;
        effective_d[k] = std::min({d_next,d_rev})/2 ;//for now I need to think more about the Dc
    }
    //*************** */
        int numRows = 100000;
        int numCols = 500;
        std::vector<int> flattened(numRows * numCols);
        for (int i = 0; i < numRows; i++) {
            for (int j = 0; j < numCols; j++) {

                flattened[i * numCols + j] = externa_polygonX[layout.graph.node_map_r[i]][j];
                
                if(externa_polygonX[layout.graph.node_map_r[i]][j]>0)
                external_edge_existtence=true;
            }
        }        
        float externa_polygonX_x[500];
        float externa_polygonX_y[500];
        scaled=scale;
        maximum_aread=maximum_area;
        
       for(int i=0;i<500;i++)
       {
            externa_polygonX_x[i]=dataStore.polygonCenters[i].first;
            externa_polygonX_y[i]=dataStore.polygonCenters[i].second;
            
    
       }

      std::cout<<"number of the polygons**"<<num_polygons<<std::endl;
        cudaCatchError(hipMalloc((void**)&d_externa_polygonX_x, 500 * sizeof(float)));
        hipMemset(d_externa_polygonX_x, 0, 500 * sizeof(float));
        cudaCatchError(hipMemcpy(d_externa_polygonX_x, externa_polygonX_x, 500 * sizeof(float), hipMemcpyHostToDevice));
        
        cudaCatchError(hipMalloc((void**)&d_externa_polygonX_y, 500 * sizeof(float)));
        hipMemset(d_externa_polygonX_y, 0, 500 * sizeof(float));
        cudaCatchError(hipMemcpy(d_externa_polygonX_y, externa_polygonX_y, 500 * sizeof(float), hipMemcpyHostToDevice));

        cudaCatchError(hipMalloc((void**)&d_externa_polygonX, numRows * numCols * sizeof(int)));
        hipMemset(d_externa_polygonX, 0, numRows * numCols * sizeof(float));
        cudaCatchError(hipMemcpy(d_externa_polygonX, flattened.data(), numRows * numCols * sizeof(int), hipMemcpyHostToDevice));
        
        cudaCatchError(hipMalloc((void**)&d_effective_d, num_points * sizeof(float)));
        hipMemset(d_effective_d, 0, num_points * sizeof(float));
        cudaCatchError(hipMemcpy(d_effective_d, effective_d, num_points * sizeof(float), hipMemcpyHostToDevice));
        
        cudaCatchError(hipMalloc((void**)&d_points, num_points*2 * sizeof(float)));
        hipMemset(d_points, 0, num_points*2 * sizeof(float));
        cudaCatchError(hipMemcpy(d_points, h_points, num_points*2 * sizeof(float), hipMemcpyHostToDevice));



      

        max_degree=0;

        for (nid_t n = 0; n < layout.graph.num_nodes(); ++n)
        {
            body_pos[n] = {layout.getX(n), layout.getY(n)};
            body_mass[n] = ForceAtlas2::mass(n);
      
            if( body_mass[n] >max_degree)
            max_degree= body_mass[n] ;
            fx[n] = 0.0;
            fy[n] = 0.0;
            fx_prev[n] = 0.0;
            fy_prev[n] = 0.0;
        }
printf("MAX DEGREE IS %d\n", max_degree);
        int cur_sources_idx = 0;
        int cur_targets_idx = 0;

        // Initialize the sources and targets arrays with edge-data.
        for (nid_t source_id = 0; source_id < layout.graph.num_nodes(); ++source_id)
        {
            for (nid_t target_id : layout.graph.neighbors_with_geq_id(source_id))
            {
                sources[cur_sources_idx++] = source_id;
                targets[cur_targets_idx++] = target_id;
            }
        }

        // GPU initialization and setup //
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        if (deviceProp.warpSize != WARPSIZE)
        {
            printf("Warpsize of device is %d, but we anticipated %d\n", deviceProp.warpSize, WARPSIZE);
            exit(EXIT_FAILURE);

        }
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
#if __CUDA_ARCH__ < 300
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
#endif
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(DisplacementKernel), hipFuncCachePreferL1);

        hipGetLastError();  // reset error value

        // Allocate space on device.
        mp_count = deviceProp.multiProcessorCount;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;

       nnodes = std::max(nbodies * 4, mp_count * max_threads_per_block * 2);

// Ensure multiple of WARPSIZE
while ((nnodes & (WARPSIZE - 1)) != 0) nnodes++;

        // child stores structure of the quadtree. values point to IDs.
        cudaCatchError(hipMalloc((void **)&childl,  sizeof(int)   * (nnodes+1) * 4));

        // the following properties, for each node in the quadtree (both internal and leaf)
        cudaCatchError(hipMalloc((void **)&body_massl,   sizeof(float) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_massl,   sizeof(float) * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&body_posl,sizeof(float2) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_posl,    sizeof(float2) * (nnodes+1)));
        // count contains the number of nested nodes for each node in quadtree
        cudaCatchError(hipMalloc((void **)&countl,  sizeof(int)   * (nnodes+1)));
        // start contains ...
        cudaCatchError(hipMalloc((void **)&startl,  sizeof(int)   * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&sortl,   sizeof(int)   * (nnodes+1)));


        cudaCatchError(hipMalloc((void **)&sourcesl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&targetsl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&fxl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fyl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fx_prevl,sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fy_prevl,sizeof(float) * (nbodies)));

        // Used for reduction in BoundingBoxKernel
        cudaCatchError(hipMalloc((void **)&maxxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&maxyl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minyl,   sizeof(float) * mp_count * FACTOR1));

        // Used for reduction in SpeedKernel
        cudaCatchError(hipMalloc((void **)&swgl,    sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&etral,   sizeof(float) * mp_count * FACTOR1));

        // Copy host data to device.
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(body_posl,  body_pos,  sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));

        // cpy fx, fy , fx_prevl, fy_prevl so they are all initialized to 0 in device memory.
        cudaCatchError(hipMemcpy(fxl, fx,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fyl, fy,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fx_prevl, fx_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fy_prevl, fy_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    }

    void CUDAForceAtlas2::freeGPUMemory()
    {
        hipDeviceSynchronize();
        hipFree(childl);

        hipFree(body_massl);
        hipFree(node_massl);
        hipFree(body_posl);
        hipFree(node_posl);
        hipFree(sourcesl);
        hipFree(targetsl);
        hipFree(countl);
        hipFree(startl);
        hipFree(sortl);

        hipFree(fxl);
        hipFree(fx_prevl);
        hipFree(fyl);
        hipFree(fy_prevl);
        hipFree(fy_prevl);


        hipFree(maxxl);
        hipFree(maxyl);
        hipFree(minxl);
        hipFree(minyl);

        hipFree(swgl);
        hipFree(etral);
        hipFree(d_points);
        hipFree(d_externa_polygonX);
        hipFree(d_externa_polygonX_x);
        hipFree(d_externa_polygonX_y);
        hipFree(d_effective_d);
        

   
    }

    CUDAForceAtlas2::~CUDAForceAtlas2()
    {

      
        free(body_mass);
        free(body_pos);
        free(sources);
        free(targets);
        free(fx);
        free(fy);
        free(fx_prev);
        free(fy_prev);
        free(h_points);

    
    }

 void CUDAForceAtlas2::doStep(int inter)
{
  ResetGlobalStateKernel<<<1, 1>>>();
cudaCatchError(hipGetLastError());
    hipGetLastError(); // clear any errors
        exGravityKernel2<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g,strong_gravity, body_massl, body_posl,fxl, fyl,Cx,  Cy,  d_points, d_externa_polygonX,d_externa_polygonX_x,  d_externa_polygonX_y,max_degree,  num_polygons,  num_points, d_effective_d);
        cudaCatchError(hipGetLastError());
         cornerKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g,strong_gravity, body_massl, body_posl,fxl, fyl,Cx,  Cy,  d_points, d_externa_polygonX,d_externa_polygonX_x,  d_externa_polygonX_y,max_degree,  num_polygons,  num_points, d_effective_d,max_distance_corner);
        cudaCatchError(hipGetLastError());
        GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, body_massl, body_posl, fxl, fyl, Cx, Cy,d_points,d_externa_polygonX,d_externa_polygonX_x,d_externa_polygonX_y,max_degree,k_r,num_points);
        cudaCatchError(hipGetLastError());
        
       
       AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, nbodies,body_massl, maximum_aread, body_posl, fxl, fyl, sourcesl, targetsl,scaled);
        cudaCatchError(hipGetLastError());
         AttractiveForceKernel2<<<mp_count * FACTOR6, THREADS6>>>(nedges, nbodies,body_massl, maximum_aread, body_posl, fxl, fyl, sourcesl, targetsl,scaled,d_externa_polygonX,d_externa_polygonX_x,d_externa_polygonX_y);
        cudaCatchError(hipGetLastError());
        BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, node_massl, body_posl, node_posl, maxxl, maxyl, minxl, minyl);
        cudaCatchError(hipGetLastError());
		
        // Build Barnes-Hut Tree
        ClearKernel1<<<mp_count, 1024>>>(nnodes, nbodies, childl);
        cudaCatchError(hipGetLastError());
        TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, body_posl, node_posl);
        cudaCatchError(hipGetLastError());
        ClearKernel2<<<mp_count, 1024>>>(nnodes, startl, node_massl);
        cudaCatchError(hipGetLastError());
        SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, body_massl, node_massl, body_posl, node_posl);
        cudaCatchError(hipGetLastError());
        SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);
        cudaCatchError(hipGetLastError());
        ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, body_massl, node_massl, body_posl, node_posl, fxl, fyl, k_r);
        cudaCatchError(hipGetLastError());
   
                
       
        

        SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, body_massl, swgl, etral);
        cudaCatchError(hipGetLastError());

        DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, body_posl, fxl, fyl, fx_prevl, fy_prevl,d_points);
        cudaCatchError(hipGetLastError());

    hipDeviceSynchronize();
    
     
}



    void CUDAForceAtlas2::retrieveLayoutFromGPU()
    {
        cudaCatchError(hipMemcpy(body_pos, body_posl, sizeof(float2) * nbodies, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendLayoutToGPU()
    {
        cudaCatchError(hipMemcpy(body_posl, body_pos, sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendGraphToGPU()
    {
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));
        //*************************
 


//**********************
        hipDeviceSynchronize();
    }

    bool CUDAForceAtlas2::sync_layout()
    {
        bool return_flag=true;
        retrieveLayoutFromGPU();
       		 for(nid_t n = 0; n < layout.graph.num_nodes(); ++n)
       	 {
            bool result=layout.move_node(n,body_pos[n].x,body_pos[n].y);
       	 	if(result==false)
            {
       	 		return_flag= false;
                
            		//layout.setX(n, body_pos[n].x);
            		//layout.setY(n, body_pos[n].y);
            }
       	 }

        	return return_flag;
    }
     void CUDAForceAtlas2::sync_layout_void()
    {
        retrieveLayoutFromGPU();
       		 for(nid_t n = 0; n < layout.graph.num_nodes(); ++n)
       	 {
       	 			//layout.move_node_void(n,body_pos[n].x,body_pos[n].y);
       	 					
            		layout.setX(n, body_pos[n].x);
            		layout.setY(n, body_pos[n].y);
       	 }
        	
    }
}
